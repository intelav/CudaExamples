#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <algorithm>
#include <cstdlib>
#include <sm_60_atomic_functions.h>
#include <stdio.h>
#include <stdlib.h>


//using namespace cooperative_groups;
namespace cg= cooperative_groups;

__device__ int reduce_sum(cooperative_groups::__v1::thread_group g,int *temp, int val){

    int lane = g.thread_rank();

    for(int i = g.size()/2;i > 0; i /=2){

        temp[lane] = val;
        g.sync();
        if(lane < i) val+= temp[lane+i];
        g.sync();
    }
    //printf("val returned in reduced_sum =%d\n",val);
    return val;
}

__device__ int thread_sum(int *input,int n){
    int sum = 0;

    for(int i = blockIdx.x * blockDim.x + threadIdx.x;
    i < n/4; i += blockDim.x * gridDim.x){
        int4 in = ((int4 *)input)[i];
        sum += in.x + in.y + in.z + in.w;

    }
    //printf("sum returned from thrad_sum =%d\n",sum);
    return sum;

}

__global__ void sum_kernel_block(int *sum,int *input,int n ){
    int my_sum = thread_sum(input, n);

    extern __shared__ int temp[];
    auto g = cooperative_groups::__v1::this_thread_block();
    int block_sum = reduce_sum(g,temp,my_sum);
    //printf("value of sum in sum_kernel_block=%d\n",*sum);
    if(g.thread_rank() == 0) 
        atomicAdd_block(sum,block_sum);

}

int main(void){
int n = 1<<24;
int blockSize = 256;
int nBlocks = (n+blockSize-1)/blockSize;
int sharedBytes = blockSize * sizeof(int);
int result;
int *sum, *data;

hipMallocManaged(&sum,sizeof(int));
hipMallocManaged(&data,n*sizeof(int));
std::fill_n(data,n,rand());
hipMemset(sum, 0, sizeof(int));

sum_kernel_block <<<nBlocks,blockSize,sharedBytes>>>(sum, data, n);
hipMemcpy(&result,sum,sizeof(int),hipMemcpyDeviceToHost);
printf("sum of 16M array number=%d\n",result);

hipFree(sum);
hipFree(data);
}